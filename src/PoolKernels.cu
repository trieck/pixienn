#include "hip/hip_runtime.h"
/********************************************************************************
* Copyright 2023 Maxar Technologies Inc.
* Author: Thomas A. Rieck
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*    http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*
* SBIR DATA RIGHTS
* Contract No. HM0476-16-C-0022
* Contractor Name: Radiant Analytic Solutions Inc.
* Contractor Address: 2325 Dulles Corner Blvd. STE 1000, Herndon VA 20171
* Expiration of SBIR Data Rights Period: 2/13/2029
*
* The Government's rights to use, modify, reproduce, release, perform, display,
* or disclose technical data or computer software marked with this legend are
* restricted during the period shown as provided in paragraph (b)(4) of the
* Rights in Noncommercial Technical Data and Computer Software-Small Business
* Innovation Research (SBIR) Program clause contained in the above identified
* contract. No restrictions apply after the expiration date shown above. Any
* reproduction of technical data, computer software, or portions thereof marked
* with this legend must also reproduce the markings.
********************************************************************************/

#include "PoolKernels.cuh"
#include "CudaUtils.cuh"
#include "CudaError.h"

namespace px {

__global__ void maxpool_kernel(int n, int in_h, int in_w, int in_c, int stride, int kernel, int pad, const float* input,
                               float* output)
{
    int h = (in_h + pad - kernel) / stride + 1;
    int w = (in_w + pad - kernel) / stride + 1;
    int c = in_c;

    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int j = id % w;
    id /= w;
    int i = id % h;
    id /= h;
    int k = id % c;
    id /= c;
    int b = id;

    int w_offset = -pad / 2;
    int h_offset = -pad / 2;

    int out_index = j + w * (i + h * (k + c * b));
    float max = -INFINITY;

    int l, m;
    for (l = 0; l < kernel; ++l) {
        for (m = 0; m < kernel; ++m) {
            int cur_h = h_offset + i * stride + l;
            int cur_w = w_offset + j * stride + m;
            int index = cur_w + in_w * (cur_h + in_h * (k + b * in_c));
            int valid = (cur_h >= 0 && cur_h < in_h &&
                         cur_w >= 0 && cur_w < in_w);
            float val = (valid != 0) ? input[index] : -INFINITY;
            max = (val > max) ? val : max;
        }
    }
    output[out_index] = max;
}

void maxpool_gpu(int n, int h, int w, int c, int stride, int kernel, int pad, const float* input, float* output)
{
    maxpool_kernel<<<cuda_gridsize(n), CUDA_BLOCK_SIZE>>>(n, h, w, c, stride, kernel, pad, input, output);

    PX_CUDA_CHECK_LAST();

}

}