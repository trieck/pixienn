#include "hip/hip_runtime.h"
/********************************************************************************
* Copyright 2020 Thomas A. Rieck, All Rights Reserved
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*    http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
********************************************************************************/

#include "CudaUtils.cuh"
#include <hip/hip_runtime.h>

#include <thrust/device_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/random.h>
#include <thrust/transform.h>

namespace px {

dim3 cuda_gridsize(std::uint32_t n)
{
    std::uint32_t k = (n - 1) / CUDA_BLOCK_SIZE + 1;
    std::uint32_t x = k;
    std::uint32_t y = 1;

    if (x > 65535) {
        x = ceil(sqrt(k));
        y = (n - 1) / (x * CUDA_BLOCK_SIZE) + 1;
    }

    dim3 d = { x, y, 1 };

    return d;
}

struct random_generator
{
    __host__ __device__ explicit random_generator(float a = 0.f, float b = 1.f);
    __host__ __device__ float operator()(std::size_t n) const;

    float a_, b_;
};

__host__ __device__ random_generator::random_generator(float a, float b) : a_(a), b_(b)
{
}

__host__ __device__ float random_generator::operator()(std::size_t n) const
{
    thrust::default_random_engine rng;
    thrust::uniform_real_distribution<float> dist(a_, b_);
    rng.discard(n);

    return dist(rng);
}

void fill_gpu(float* ptr, std::size_t n, float value)
{
    thrust::device_ptr<float> dev_ptr = thrust::device_pointer_cast(ptr);
    thrust::fill(dev_ptr, dev_ptr + n, value);
}

void random_generate_gpu(float* ptr, std::size_t n, float a, float b)
{
    thrust::device_ptr<float> dev_ptr = thrust::device_pointer_cast(ptr);

    thrust::counting_iterator<unsigned int> index_sequence_begin(0);
    thrust::transform(index_sequence_begin,
                      index_sequence_begin + n,
                      dev_ptr,
                      random_generator(a, b));
}

}   // px